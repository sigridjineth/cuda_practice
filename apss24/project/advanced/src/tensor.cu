#include "tensor.h"
#include <cstring>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

Tensor::Tensor(const vector<size_t> &shape_, size_t batch_size) {
    shape = shape_;
    ndim = shape.size();
    if (ndim == 0) {
        CHECK_CUDA(hipHostMalloc(&buf, batch_size * sizeof(half_cpu)));
        CHECK_CUDA(hipMalloc(&d_buf, batch_size * sizeof(half)));
    } else {
        size_t N = num_elem() * batch_size;
        CHECK_CUDA(hipHostMalloc(&buf, N * sizeof(half_cpu)));
        CHECK_CUDA(hipMalloc(&d_buf, N * sizeof(half)));
    }
}

Tensor::Tensor(const vector<size_t> &shape_, half_cpu *data) {
    shape = shape_;
    ndim = shape.size();
    size_t N = num_elem();
    CHECK_CUDA(hipHostMalloc(&buf, N * sizeof(half_cpu)));
    memcpy(buf, data, N * sizeof(half_cpu));
    CHECK_CUDA(hipMalloc(&d_buf, N * sizeof(half)));
    to_device(1); // batch_size를 1로 설정
}

Tensor::~Tensor() {
    if (buf != nullptr) CHECK_CUDA(hipHostFree(buf));
    if (d_buf != nullptr) CHECK_CUDA(hipFree(d_buf));
}

size_t Tensor::num_elem() {
    size_t size = 1;
    for (size_t i = 0; i < ndim; i++) { size *= shape[i]; }
    return size;
}

void Tensor::to_device(size_t batch_size) {
    size_t N = num_elem() * batch_size;
    CHECK_CUDA(hipMemcpy(d_buf, buf, N * sizeof(half), hipMemcpyHostToDevice));
}

void Tensor::to_host(size_t batch_size) {
    size_t N = num_elem() * batch_size;
    CHECK_CUDA(hipMemcpy(buf, d_buf, N * sizeof(half), hipMemcpyDeviceToHost));
}

void Tensor::to_device_async(hipStream_t stream, size_t batch_size) {
    size_t N = num_elem() * batch_size;
    CHECK_CUDA(hipMemcpyAsync(d_buf, buf, N * sizeof(half), hipMemcpyHostToDevice, stream));
}

void Tensor::to_host_async(hipStream_t stream, size_t batch_size) {
    size_t N = num_elem() * batch_size;
    CHECK_CUDA(hipMemcpyAsync(buf, d_buf, N * sizeof(half), hipMemcpyDeviceToHost, stream));
}

void Tensor::resize(const vector<size_t> &new_shape, size_t batch_size) {
    shape = new_shape;
    ndim = new_shape.size();
    size_t N = num_elem() * batch_size;
    CHECK_CUDA(hipHostFree(buf));
    CHECK_CUDA(hipFree(d_buf));
    CHECK_CUDA(hipHostMalloc(&buf, N * sizeof(half_cpu)));
    CHECK_CUDA(hipMalloc(&d_buf, N * sizeof(half)));
}

void Tensor::set_data(half_cpu *data, size_t batch_size) {
    size_t N = num_elem() * batch_size;
    memcpy(buf, data, N * sizeof(half_cpu));
}

Parameter::Parameter(const vector<size_t> &shape_, half_cpu *data) : Tensor(shape_, 1) {
    size_t N = num_elem();
    memcpy(buf, data, N * sizeof(half_cpu));
    to_device(1);
}