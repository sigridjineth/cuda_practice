#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>

#include "layer.h"
#include "model.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

/* [Model Parameters]
 * _w: Weight parameter
 * _b: Bias parameter
 */
Parameter *mlp1_w, *mlp1_b;
Parameter *mlp2_w, *mlp2_b;
Parameter *convtrans1_w, *convtrans1_b;
Parameter *batchnorm1_w, *batchnorm1_b;
Parameter *convtrans2_w, *convtrans2_b;
Parameter *batchnorm2_w, *batchnorm2_b;
Parameter *convtrans3_w, *convtrans3_b;
Parameter *batchnorm3_w, *batchnorm3_b;
Parameter *convtrans4_w, *convtrans4_b;
Parameter *batchnorm4_w, *batchnorm4_b;
Parameter *convtrans5_w, *convtrans5_b;
Parameter *batchnorm5_w, *batchnorm5_b;
Parameter *convtrans6_w, *convtrans6_b;
Parameter *batchnorm6_w, *batchnorm6_b;
Parameter *conv_w, *conv_b;

void alloc_and_set_parameters(half_cpu *param, size_t param_size) {
    size_t pos = 0;

    mlp1_w = new Parameter(vector<size_t>{16384, 128}, param + pos);
    pos += 16384 * 128;
    mlp1_b = new Parameter(vector<size_t>{16384}, param + pos);
    pos += 16384;

    mlp2_w = new Parameter(vector<size_t>{4096, 16384}, param + pos);
    pos += 4096 * 16384;
    mlp2_b = new Parameter(vector<size_t>{4096}, param + pos);
    pos += 4096;

    convtrans1_w = new Parameter(vector<size_t>{1024, 512, 3, 3}, param + pos);
    pos += 1024 * 512 * 3 * 3;
    convtrans1_b = new Parameter(vector<size_t>{512}, param + pos);
    pos += 512;
    batchnorm1_w = new Parameter(vector<size_t>{512}, param + pos);
    pos += 512;
    batchnorm1_b = new Parameter(vector<size_t>{512}, param + pos);
    pos += 512;

    convtrans2_w = new Parameter(vector<size_t>{512, 256, 3, 3}, param + pos);
    pos += 512 * 256 * 3 * 3;
    convtrans2_b = new Parameter(vector<size_t>{256}, param + pos);
    pos += 256;
    batchnorm2_w = new Parameter(vector<size_t>{256}, param + pos);
    pos += 256;
    batchnorm2_b = new Parameter(vector<size_t>{256}, param + pos);
    pos += 256;

    convtrans3_w = new Parameter(vector<size_t>{256, 128, 3, 3}, param + pos);
    pos += 256 * 128 * 3 * 3;
    convtrans3_b = new Parameter(vector<size_t>{128}, param + pos);
    pos += 128;
    batchnorm3_w = new Parameter(vector<size_t>{128}, param + pos);
    pos += 128;
    batchnorm3_b = new Parameter(vector<size_t>{128}, param + pos);
    pos += 128;

    convtrans4_w = new Parameter(vector<size_t>{128, 64, 3, 3}, param + pos);
    pos += 128 * 64 * 3 * 3;
    convtrans4_b = new Parameter(vector<size_t>{64}, param + pos);
    pos += 64;
    batchnorm4_w = new Parameter(vector<size_t>{64}, param + pos);
    pos += 64;
    batchnorm4_b = new Parameter(vector<size_t>{64}, param + pos);
    pos += 64;

    convtrans5_w = new Parameter(vector<size_t>{64, 32, 3, 3}, param + pos);
    pos += 64 * 32 * 3 * 3;
    convtrans5_b = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;
    batchnorm5_w = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;
    batchnorm5_b = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;

    convtrans6_w = new Parameter(vector<size_t>{32, 32, 3, 3}, param + pos);
    pos += 32 * 32 * 3 * 3;
    convtrans6_b = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;
    batchnorm6_w = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;
    batchnorm6_b = new Parameter(vector<size_t>{32}, param + pos);
    pos += 32;

    conv_w = new Parameter(vector<size_t>{3, 32, 3, 3}, param + pos);
    pos += 3 * 32 * 3 * 3;
    conv_b = new Parameter(vector<size_t>{3}, param + pos);
    pos += 3;

    if (pos != param_size) {
        fprintf(stderr, "Parameter size mismatched: %zu vs %zu\n", pos, param_size);
        exit(1);
    }
}

void free_parameters() {
    delete mlp1_w;
    delete mlp1_b;
    delete mlp2_w;
    delete mlp2_b;
    delete convtrans1_w;
    delete convtrans1_b;
    delete batchnorm1_w;
    delete batchnorm1_b;
    delete convtrans2_w;
    delete convtrans2_b;
    delete batchnorm2_w;
    delete batchnorm2_b;
    delete convtrans3_w;
    delete convtrans3_b;
    delete batchnorm3_w;
    delete batchnorm3_b;
    delete convtrans4_w;
    delete convtrans4_b;
    delete batchnorm4_w;
    delete batchnorm4_b;
    delete convtrans5_w;
    delete convtrans5_b;
    delete batchnorm5_w;
    delete batchnorm5_b;
    delete convtrans6_w;
    delete convtrans6_b;
    delete batchnorm6_w;
    delete batchnorm6_b;
    delete conv_w;
    delete conv_b;
}

/* [Model Activations]
 * _a: Activation buffer
 */
Activation *linear1_a, *linear2_a;
Activation *reshape_a;
Activation *convtrans1_a, *batchnorm1_a;
Activation *convtrans2_a, *batchnorm2_a;
Activation *convtrans3_a, *batchnorm3_a;
Activation *convtrans4_a, *batchnorm4_a;
Activation *convtrans5_a, *batchnorm5_a;
Activation *convtrans6_a, *batchnorm6_a;
Activation *conv_a;

void alloc_activations(size_t batch_size) {
    linear1_a = new Activation(vector<size_t>{batch_size, 16384});
    linear2_a = new Activation(vector<size_t>{batch_size, 4096});
    reshape_a = new Activation(vector<size_t>{batch_size, 1024, 2, 2});
    convtrans1_a = new Activation(vector<size_t>{batch_size, 512, 4, 4});
    batchnorm1_a = new Activation(vector<size_t>{batch_size, 512, 4, 4});
    convtrans2_a = new Activation(vector<size_t>{batch_size, 256, 8, 8});
    batchnorm2_a = new Activation(vector<size_t>{batch_size, 256, 8, 8});
    convtrans3_a = new Activation(vector<size_t>{batch_size, 128, 16, 16});
    batchnorm3_a = new Activation(vector<size_t>{batch_size, 128, 16, 16});
    convtrans4_a = new Activation(vector<size_t>{batch_size, 64, 32, 32});
    batchnorm4_a = new Activation(vector<size_t>{batch_size, 64, 32, 32});
    convtrans5_a = new Activation(vector<size_t>{batch_size, 32, 64, 64});
    batchnorm5_a = new Activation(vector<size_t>{batch_size, 32, 64, 64});
    convtrans6_a = new Activation(vector<size_t>{batch_size, 32, 128, 128});
    batchnorm6_a = new Activation(vector<size_t>{batch_size, 32, 128, 128});
    conv_a = new Activation(vector<size_t>{batch_size, 3, 128, 128});
}

void free_activations() {
    delete linear1_a;
    delete linear2_a;
    delete reshape_a;
    delete convtrans1_a;
    delete batchnorm1_a;
    delete convtrans2_a;
    delete batchnorm2_a;
    delete convtrans3_a;
    delete batchnorm3_a;
    delete convtrans4_a;
    delete batchnorm4_a;
    delete convtrans5_a;
    delete batchnorm5_a;
    delete convtrans6_a;
    delete batchnorm6_a;
    delete conv_a;
}

void generate_images(half_cpu *input, half_cpu *output, size_t n_img) {
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    Tensor *z = new Tensor(vector<size_t>{LATENT_DIM}, n_img);
    z->set_data(input, n_img);
    z->to_device_async(stream, n_img);

    alloc_activations(n_img);

    Linear(z, mlp1_w, mlp1_b, linear1_a, stream, n_img);
    Linear(linear1_a, mlp2_w, mlp2_b, linear2_a, stream, n_img);
    Reshape(linear2_a, reshape_a, stream, n_img);

    ConvTranspose2d(reshape_a, convtrans1_w, convtrans1_b, convtrans1_a, stream, n_img);
    BatchNorm2d(convtrans1_a, batchnorm1_w, batchnorm1_b, batchnorm1_a, stream, n_img);
    LeakyReLU(batchnorm1_a, stream, n_img);

    ConvTranspose2d(batchnorm1_a, convtrans2_w, convtrans2_b, convtrans2_a, stream, n_img);
    BatchNorm2d(convtrans2_a, batchnorm2_w, batchnorm2_b, batchnorm2_a, stream, n_img);
    LeakyReLU(batchnorm2_a, stream, n_img);

    ConvTranspose2d(batchnorm2_a, convtrans3_w, convtrans3_b, convtrans3_a, stream, n_img);
    BatchNorm2d(convtrans3_a, batchnorm3_w, batchnorm3_b, batchnorm3_a, stream, n_img);
    LeakyReLU(batchnorm3_a, stream, n_img);

    ConvTranspose2d(batchnorm3_a, convtrans4_w, convtrans4_b, convtrans4_a, stream, n_img);
    BatchNorm2d(convtrans4_a, batchnorm4_w, batchnorm4_b, batchnorm4_a, stream, n_img);
    LeakyReLU(batchnorm4_a, stream, n_img);

    ConvTranspose2d(batchnorm4_a, convtrans5_w, convtrans5_b, convtrans5_a, stream, n_img);
    BatchNorm2d(convtrans5_a, batchnorm5_w, batchnorm5_b, batchnorm5_a, stream, n_img);
    LeakyReLU(batchnorm5_a, stream, n_img);

    ConvTranspose2d(batchnorm5_a, convtrans6_w, convtrans6_b, convtrans6_a, stream, n_img);
    BatchNorm2d(convtrans6_a, batchnorm6_w, batchnorm6_b, batchnorm6_a, stream, n_img);
    LeakyReLU(batchnorm6_a, stream, n_img);

    Conv2d(batchnorm6_a, conv_w, conv_b, conv_a, stream, n_img);
    Tanh(conv_a, stream, n_img);

    conv_a->to_host_async(stream, n_img);

    CHECK_CUDA(hipStreamSynchronize(stream));

    memcpy(output, conv_a->buf, n_img * 3 * 128 * 128 * sizeof(half_cpu));

    CHECK_CUDA(hipStreamDestroy(stream));

    delete z;
    free_activations();
}